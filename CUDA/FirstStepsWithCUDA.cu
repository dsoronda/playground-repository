#include "hip/hip_runtime.h"
#include <stdio.h>
#include <helper_timer.h>

#include "hip/hip_runtime.h"
#include ""

__global__ void addKernel(const int *a, const int *b, int *c, int size) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;

  float _a = a[i];
  float _b = b[i];

  float sum = _a + _b * 2.0f;
  float mul = _a * (_a + _b);
  float val = (sum + mul) * 5.35f;
  float val2= (sum * 8.06f) + sqrtf(val) * (mul + 6.36f);

  c[i] = sqrt(val2) * 44.87f;
}

void task4() {
  const int VALUE = 100;
  int someConstant = 1024;
  int size = 0;
  int block = 0;

  int *a = 0;
  int *b = 0;
  int *c = 0;

  int i = 0;
  int j = 0;

  int sum = 0;
  int sum2 = 0;

  int *aOnDevice = 0;
  int *bOnDevice = 0;
  int *cOnDevice = 0;

  do {
    printf("Type vector size: ");
    scanf("%d", &size);

    if (size <= 0) break;

    printf("\nType size of block: ");
    scanf("%d", &someConstant);

    if (size % someConstant == 0) {
      block = size / someConstant;
    } else {
      block = size / someConstant + 1;
    }

    a = (int*)malloc(sizeof(int) * size);
    b = (int*)malloc(sizeof(int) * size);
    c = (int*)malloc(sizeof(int) * size);

    for(i = 0; i < size; ++i) {
      a[i] = 1;
      b[i] = 1;
    }

    hipSetDevice(0);

    hipMalloc((void**)&aOnDevice, size * sizeof(int));
    hipMalloc((void**)&bOnDevice, size * sizeof(int));
    hipMalloc((void**)&cOnDevice, size * sizeof(int));

    hipMemcpy(aOnDevice, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(bOnDevice, b, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(cOnDevice, c, size * sizeof(int), hipMemcpyHostToDevice);

    dim3 gridDims = dim3(block, 1, 1);
    dim3 blockDims = dim3(someConstant, 1, 1);

    StopWatchInterface* hTimer;
    sdkCreateTimer(&hTimer);
    sdkResetTimer(&hTimer);

    hipDeviceSynchronize();
    addKernel<<<gridDims,blockDims>>>(aOnDevice, bOnDevice, cOnDevice, size);

    sdkStartTimer(&hTimer);
    for(i = 0; i < VALUE; ++i) {
      addKernel<<<gridDims,blockDims>>>(aOnDevice, bOnDevice, cOnDevice, size);
    }
    hipDeviceSynchronize();
    sdkStopTimer(&hTimer);

    hipMemcpy(c, cOnDevice, size * sizeof(int), hipMemcpyDeviceToHost);
    float time1 = sdkGetTimerValue(&hTimer) / VALUE;

    sdkResetTimer(&hTimer);

    hipFree(aOnDevice);
    hipFree(bOnDevice);

    sum = c[0];
    for(i = 1; i < size; ++i) {
      sum += c[i];
    }

    sdkStartTimer(&hTimer);

    for(i = 0; i < size; ++i)
    {
      float _a = a[i];
      float _b = b[i];
      float sum = _a + _b * 2.0f;
      float mul = _a * (_a + _b);
      float val = (sum + mul) * 5.35f;
      float val2 = (sum * 8.06f) + sqrtf(val) * (mul + 6.36f);
      c[i] = sqrt(val2) * 44.87f;
    }

    sdkStopTimer(&hTimer);
    float time2 = sdkGetTimerValue(&hTimer);

    sum2 = 0;

    for(i = 0; i < size; ++i) {
      sum2 += c[i];
    }

    if (sum == sum2) {
      printf("OK!\n");
    }

    printf("CUDA: %f, CPU: %f \n", time1, time2);

    sdkResetTimer(&hTimer);
    hipDeviceReset();

    free(a);
    free(b);
    free(c);
  } while(size > 0);
}