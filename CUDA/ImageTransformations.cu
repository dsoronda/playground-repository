#include "hip/hip_runtime.h"
#include <stdio.h>
#include <helper_timer.h>
#include <helper_image.h>

#include "hip/hip_runtime.h"
#include ""

__global__ void imageKernel(const float *a, float *b, int im_width, int im_height, int pitch) {
  int ix = threadIdx.x + blockDim.x * blockIdx.x;
  int iy = threadIdx.y + blockDim.y * blockIdx.y;
  float sum = 0.0f;
  int oxx, oyy;

  for (int xx = ix - 2; xx < ix + 2; ++xx) {
    for (int yy = iy - 2; yy < iy + 2; ++yy) {
      oxx = xx;
      oyy = yy;

      if (xx < 0) oxx = 0;
      if (yy < 0) oyy = 0;

      if (xx >= im_width) oxx = im_width - 1;
      if (yy >= im_height) oyy = im_height - 1;

      sum += a[oyy * (pitch / sizeof(float)) + oxx];
    }
  }

  if (ix < im_width && iy < im_height) {
    int idx = iy * (pitch / sizeof(float)) + ix;
    b[idx] = sum / 25;

    if (b[idx] > 1.0f) {
      b[idx] = 1.0f;
    }

    if (b[idx] < 0.0f) {
      b[idx] = 0.0f;
    }
  }
}

void blurPixel(const float* a, float* b, int ix, int iy, int im_width, int im_height) {
  float sum = 0.0f;
  int oxx, oyy;

  for (int xx = ix - 2; xx < ix + 2; ++xx) {
    for (int yy = iy - 2; yy < iy + 2; ++yy) {
      oxx = xx;
      oyy = yy;

      if (xx < 0) oxx = 0;
      if (yy < 0) oyy = 0;

      if (xx >= im_width) oxx = im_width - 1;
      if (yy >= im_height) oyy = im_height - 1;

      sum += a[oyy * im_width + oxx];
    }
  }

  if (ix < im_width && iy < im_height) {
    int idx = iy * im_width + ix;
    b[idx] = sum / 25;

    if (b[idx] > 1.0f) {
      b[idx] = 1.0f;
    }

    if (b[idx] < 0.0f) {
      b[idx] = 0.0f;
    }
  }
}

void task4() {
  int blockSizeX = 0, blockSizeY = 0;
  int size = 0;
  int blockX = 0, blockY = 0;

  float *b = 0;
  float *c = 0;
  float *d = 0;

  size_t pitch = 0;
  size_t pitch1 = 0;

    float *aOnDevice = 0;
    float *bOnDevice = 0;

  unsigned int im_width, im_height;
  float* im_data = NULL;

  sdkLoadPGM("lena.pgm", &im_data, &im_width, &im_height);
  size = im_width * im_height;

  printf("Type size of block X: ");
  scanf("%d", &blockSizeX);

  printf("\nType size of Y block: ");
  scanf("%d", &blockSizeY);

  if (im_width % blockSizeX == 0) {
    blockX = im_width / blockSizeX;
  } else {
    blockX = im_width / blockSizeX + 1;
  }

  if (im_height % blockSizeY == 0) {
    blockY = im_height / blockSizeY;
  } else {
    blockY = im_height / blockSizeY + 1;
  }

  b = (float*)malloc(sizeof(float) * size);
  c = (float*)malloc(sizeof(float) * size);
  d = (float*)malloc(sizeof(float) * size);

  hipSetDevice(0);
  hipMallocPitch((void**)&aOnDevice, &pitch, im_width * sizeof(float), im_height);
  hipMallocPitch((void**)&bOnDevice, &pitch1, im_width * sizeof(float), im_height);

  hipMemcpy2D(aOnDevice, pitch, im_data, im_width * sizeof(float), im_width * sizeof(float), im_height, hipMemcpyHostToDevice);
  hipMemcpy2D(bOnDevice, pitch1, im_data, im_width * sizeof(float), im_width * sizeof(float), im_height, hipMemcpyHostToDevice);

  dim3 gridDims = dim3(blockX, blockY, 1);
  dim3 blockDims = dim3(blockSizeX, blockSizeY, 1);

  StopWatchInterface* hTimer;
  sdkCreateTimer(&hTimer);
  sdkResetTimer(&hTimer);

  hipDeviceSynchronize();

  sdkStartTimer(&hTimer);

  for(int i = 0; i < 100; ++i) {
    imageKernel<<<gridDims,blockDims>>>(aOnDevice, bOnDevice, im_width, im_height, pitch);
  }

  hipDeviceSynchronize();
  sdkStopTimer(&hTimer);

  float time1 = sdkGetTimerValue(&hTimer) / 100;

  hipMemcpy2D(c, im_width * sizeof(float), bOnDevice, pitch1, im_width * sizeof(float), im_height, hipMemcpyDeviceToHost);
  sdkSavePGM("lena_out_gpu.pgm", c, im_width, im_height);

  hipFree(aOnDevice);
  hipFree(bOnDevice);

  sdkResetTimer(&hTimer);

  sdkStartTimer(&hTimer);

  for(int x = 0; x < im_width; ++x) {
    for(int y = 0; y  < im_height; ++y) {
      blurPixel(im_data, d, x, y, im_width, im_height);
    }
  }

  sdkStopTimer(&hTimer);
  float time2 = sdkGetTimerValue(&hTimer);

  printf("CUDA: %f, CPU: %f \n", time1, time2);
  sdkSavePGM("lena_out_cpu.pgm", d, im_width, im_height);

  sdkResetTimer(&hTimer);
  hipDeviceReset();

  free(b);
  free(c);
  free(d);
}